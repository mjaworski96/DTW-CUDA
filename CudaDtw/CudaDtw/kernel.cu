#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;

# define DELLEXPORT extern "C" __declspec(dllexport)
typedef double SIGNAL;
#define huge 1e32

__device__
SIGNAL _min(SIGNAL a, SIGNAL b)
{
	if (a < b)
		return a;
	else
		return b;
}
__device__
SIGNAL _max(SIGNAL a, SIGNAL b)
{
	if (a > b)
		return a;
	else
		return b;
}
__device__
SIGNAL _min(SIGNAL a, SIGNAL b, SIGNAL c)
{
	return min(min(a, b), c);
}
__device__
SIGNAL _abs(SIGNAL a)
{
	if (a < 0)
		return -a;
	else
		return a;
}
__device__
SIGNAL _dist(SIGNAL a, SIGNAL b)
{
	return _abs(a - b);
}
__device__
//Matrix index to array index
int _mitai(int i, int j, int size)
{
	return i * size + j;
}
__device__
//Matrix index to buffer index
int _mitbi(int mi, int start_index)
{
	return mi - start_index;
}
__device__
SIGNAL _save_get_value(SIGNAL* buffer, int buffer_size, int index) {
	if (index < 0 || index >= buffer_size)
		return huge;
	else
		return buffer[index];
}

__device__
SIGNAL dtw_distance(SIGNAL* a, int a_i,
	SIGNAL* b, int b_i,
	int ts_size, int window)
{
	int buffers_size = 2 * window;
	SIGNAL* first = (SIGNAL*)malloc(buffers_size * sizeof(SIGNAL));
	SIGNAL* second = (SIGNAL*)malloc(buffers_size * sizeof(SIGNAL));
	memset(first, huge, buffers_size * sizeof(SIGNAL));
	memset(second, huge, buffers_size * sizeof(SIGNAL));
	int first_start_index = -window - 1;
	int second_start_index = -window;;
	for (int i = 0; i < ts_size; i++)
	{
		SIGNAL a_value = a[_mitai(a_i, i, ts_size)];
		for (int j = second_start_index; j < _min(ts_size, i + window); j++)
		{
			SIGNAL b_value = b[_mitai(b_i, j, ts_size)];
			SIGNAL dist = _dist(a_value, b_value);
			if (i == 0 && j == 0) {
				second[window] = dist;
			}
			else if (i == 0)
			{
				int second_index = window + j;
				second[second_index] = second[second_index - 1] + dist;
			}
			else if (j == 0)
			{
				second[window - i] = first[window - i + 1] + dist;
			}
			else {
				int first_index = _mitbi(j, first_start_index);
				int second_index = _mitbi(j, second_start_index);
				second[second_index] = _min(
					_save_get_value(first, buffers_size, first_index - 1),
					_save_get_value(second, buffers_size, second_index - 1),
					_save_get_value(first, buffers_size, first_index))
					+ dist;
			}
		}
		SIGNAL *tmp = first;
		first = second;
		second = tmp;

		first_start_index++;
		second_start_index++;
	}

	SIGNAL result = first[window];
	free(first);
	free(second);
	return result;
}

__global__
void cuda_dtw(SIGNAL* a, SIGNAL* b, SIGNAL* result,
	int a_size, int b_size, int ts_size, int window)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	int size = a_size * b_size;
	for (int i = index; i < size; i += stride) {
		//int start = i * ts_size;
		int a_i = i / a_size;
		int b_i = i % a_size;
		result[i] = dtw_distance(a, a_i,
			b, b_i,
			ts_size, window);
	}
}


DELLEXPORT void dtw_gpu(SIGNAL* a, SIGNAL* b, SIGNAL* result,
	int a_size, int b_size, int ts_size, int maxWindow) {

	const size_t malloc_limit = 3ull * 1024ull * 1024ull * 1024ull;
	hipDeviceSetLimit(hipLimitMallocHeapSize, malloc_limit);

	SIGNAL* gpu_a;
	SIGNAL* gpu_b;
	SIGNAL* gpu_result;
	hipMallocManaged(&gpu_a, a_size * ts_size * sizeof(SIGNAL));
	hipMallocManaged(&gpu_b, b_size * ts_size * sizeof(SIGNAL));
	hipMallocManaged(&gpu_result, a_size * b_size * sizeof(SIGNAL));

	hipMemcpy(gpu_a, a, a_size * ts_size * sizeof(SIGNAL), hipMemcpyHostToDevice);
	hipMemcpy(gpu_b, b, b_size * ts_size * sizeof(SIGNAL), hipMemcpyHostToDevice);

	int blockSize = 256;
	int numBlocks = ceil(((a_size * b_size) + blockSize - 1) / (1.0 * blockSize));

	cuda_dtw
		<< <numBlocks, blockSize >> >
		(gpu_a, gpu_b, gpu_result,
			a_size, b_size, ts_size, maxWindow
			);
	hipDeviceSynchronize();
	hipMemcpy(result, gpu_result, a_size * b_size * sizeof(SIGNAL), hipMemcpyDeviceToHost);

	hipFree(gpu_a);
	hipFree(gpu_b);
	hipFree(gpu_result);
}